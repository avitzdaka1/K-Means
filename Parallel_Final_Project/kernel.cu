#include "hip/hip_runtime.h"
#include "main_header.h"
point_t* GlobalPointsCuda = 0;

//	Kernel point relocation function
__global__ void pointRelocationKernel(point_t* pointsArrCuda, double timeInterval, double currentT, int numOfPoints, double cosT, double sinT) {	//	Kernel point relocation function using GPU.
			int id = blockIdx.x * blockDim.x + threadIdx.x;
			if (id < numOfPoints) {
				double centerX = pointsArrCuda[id].a, centerY = pointsArrCuda[id].b;		//	Saves current X, Y coordinates.
				pointsArrCuda[id].x = centerX + (pointsArrCuda[id].radius * cosT);		//	Calculates new X coordinate and stores it in the point's X value.
				pointsArrCuda[id].y = centerY + (pointsArrCuda[id].radius * sinT);		//	Calculates new Y coordinate and stores it in the point's Y value.
			}
}

//	Point relocation function using GPU.
hipError_t pointRelocationCuda(point_t* pointsArr, double timeInterval,
	double currentT, int numOfPoints, double cosT, double sinT) {
	hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
	int threadsPerBlock = deviceProp.maxThreadsPerBlock / 3;
	int numOfBlocks = (numOfPoints / threadsPerBlock) + 1;
    hipError_t cudaStatus;
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) 
		fprintf(stderr, "hipSetDevice failed! Do you have a CUDA-capable GPU installed?");
    
	// Launch a kernel on the GPU with one thread for each element.
	pointRelocationKernel<<<numOfBlocks, threadsPerBlock>>>(GlobalPointsCuda, timeInterval, currentT, numOfPoints, cosT, sinT);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) 
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
  
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) 
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(pointsArr, GlobalPointsCuda, numOfPoints * sizeof(point_t), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess)
        fprintf(stderr, "hipMemcpy failed!");

    return cudaStatus;
}

//	Allocates memory space for the points array in the GPU and copies the points array to that memory.
hipError_t allocatePointsCuda(point_t* pointsArr, int numOfPoints) {
	hipError_t cudaStatus;
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess)
        fprintf(stderr, "hipSetDevice failed! Do you have a CUDA-capable GPU installed?");

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&GlobalPointsCuda, numOfPoints * sizeof(point_t));
    if (cudaStatus != hipSuccess)
        fprintf(stderr, "hipMalloc failed!");


    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(GlobalPointsCuda, pointsArr, numOfPoints * sizeof(point_t), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) 
        fprintf(stderr, "hipMemcpy failed!");
    return cudaStatus;
}
